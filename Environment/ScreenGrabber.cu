#include "ScreenGrabber.h"
#include <GL/glew.h>
#include <iostream>
#include <vector>

#include <GL/gl.h>
#include <chrono>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <fstream>

// #define STB_IMAGE_WRITE_IMPLEMENTATION
#include "fpng.h"

namespace
{

void writePNG(const char *path, const unsigned char *rgba, int w, int h, bool flip_y = true)
{
    static bool fpng_initialized = false;
    if (!fpng_initialized)
    {
        fpng_initialized = true;
        fpng::fpng_init();
    }
    const unsigned char       *src = rgba;
    std::vector<unsigned char> tmp;
    if (flip_y)
    {
        tmp.resize(size_t(w) * h * 4);
        for (int y = 0; y < h; ++y)
            memcpy(&tmp[size_t(y) * w * 4], &rgba[size_t(h - 1 - y) * w * 4], size_t(w) * 4);
        src = tmp.data();
    }
    if (!fpng::fpng_encode_image_to_file(path, src, w, h, 4))
        std::cerr << "FPNG write failed: " << path << "\n";
}

void saveCudaArrayToFileAsPPM(hipArray_t cuda_array,
                              const int   w,
                              const int   h,
                              const char *path,
                              const bool  flip_y = true)
{
    std::vector<unsigned char> host_data(static_cast<size_t>(w * h * 4));
    const size_t               pitch{static_cast<size_t>(w) * 4}; // bytes per row on host
    if (hipMemcpy2DFromArray(host_data.data(), pitch, cuda_array, 0, 0, pitch, h, hipMemcpyDeviceToHost) ==
        hipSuccess)
    {
        writePNG(path, host_data.data(), w, h, flip_y);
    }
    else
    {
        std::cerr << "Failed to copy data from CUDA array to host memory." << std::endl;
    }
}

} // namespace

class ScreenGrabber::Impl
{

  public:
    Impl(const unsigned int id, const int width, const int height) : width_(width), height_(height)
    {
        hipGraphicsGLRegisterImage(&cuda_resource_, id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    }

    ~Impl() = default;

    void saveRenderTargetToFile(const std::string &filename)
    {
        hipGraphicsMapResources(1, &cuda_resource_, 0);
        hipArray_t cuda_array;
        hipGraphicsSubResourceGetMappedArray(&cuda_array, cuda_resource_, 0, 0);

        saveCudaArrayToFileAsPPM(cuda_array, width_, height_, filename.c_str(), true);
    }

  private:
    hipGraphicsResource_t cuda_resource_;
    const int              width_;
    const int              height_;
};

ScreenGrabber::ScreenGrabber(const unsigned int id, const int width, const int height)
    : impl_(std::make_unique<Impl>(id, width, height))
{
}

ScreenGrabber::~ScreenGrabber() = default;

void ScreenGrabber::saveRenderTargetToFile(const std::string &filename)
{
    impl_->saveRenderTargetToFile(filename);
}